
#include <hip/hip_runtime.h>
#include <vector>
// #include <cuda_runtime.h>

struct Particle {
    float2 position;  // Use float2 for simplicity
};

// GPU kernel for assigning particles to grid
__global__
void assignParticlesKernel(Particle* particles, int numParticles, int* cellCounts, Particle** cellData, 
                           int nColumns, int nRows, float cellSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    Particle& particle = particles[idx];
    int col = static_cast<int>(particle.position.x / cellSize);
    int row = static_cast<int>(particle.position.y / cellSize);

    if (col >= 0 && col < nColumns && row >= 0 && row < nRows) {
        int cellIdx = row * nColumns + col;

        // Atomic operation to add particle to cell
        int pos = atomicAdd(&cellCounts[cellIdx], 1);
        cellData[cellIdx][pos] = particle;
    }
}

// Wrapper function
void assignParticlesToGrid(std::vector<Particle>& particles, 
                           std::vector<std::vector<std::vector<Particle>>>& cells,
                           int nColumns, int nRows, float cellSize) {
    int numParticles = particles.size();
    int numCells = nColumns * nRows;

    // Flatten cells data structure for GPU use
    std::vector<int> cellCounts(numCells, 0);
    std::vector<Particle*> cellData(numCells, nullptr);

    // Allocate memory on GPU
    Particle* d_particles;
    int* d_cellCounts;
    Particle** d_cellData;

    hipMalloc(&d_particles, numParticles * sizeof(Particle));
    hipMalloc(&d_cellCounts, numCells * sizeof(int));
    hipMalloc(&d_cellData, numCells * sizeof(Particle*));

    // Allocate memory for individual cell particle pointers
    for (int i = 0; i < numCells; ++i) {
        hipMalloc(&cellData[i], particles.size() * sizeof(Particle)); // Over-allocate for simplicity
    }

    // Copy data to GPU
    hipMemcpy(d_particles, particles.data(), numParticles * sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(d_cellCounts, cellCounts.data(), numCells * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cellData, cellData.data(), numCells * sizeof(Particle*), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;
    assignParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_particles, numParticles, d_cellCounts, d_cellData,
                                                              nColumns, nRows, cellSize);

    // Copy results back to host
    hipMemcpy(cellCounts.data(), d_cellCounts, numCells * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < numCells; ++i) {
        hipMemcpy(cells[i / nColumns][i % nColumns].data(), cellData[i], cellCounts[i] * sizeof(Particle), hipMemcpyDeviceToHost);
    }

    // Free GPU memory
    hipFree(d_particles);
    hipFree(d_cellCounts);
    for (int i = 0; i < numCells; ++i) {
        hipFree(cellData[i]);
    }
    hipFree(d_cellData);
}
